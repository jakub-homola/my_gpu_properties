#include <cstdio>
#include <cstdlib>


void print_properties_for_gpu(int gpu_idx)
{
    hipDeviceProp_t p;
    hipGetDeviceProperties(&p, gpu_idx);

    printf("Device %d:\n", gpu_idx);
    printf("  Name: %s\n", p.name);
    printf("  Compute capability: %d.%d\n", p.major, p.minor);
    printf("  Multiprocesor count: %d\n", p.multiProcessorCount);
    printf("  Warp size: %d\n", p.warpSize);
    printf("  ECC memory enabled: %s\n", p.ECCEnabled ? "yes" : "no");
    printf("  Global memory: %ld MiB\n", p.totalGlobalMem >> 20);
    printf("  L2 cache: %d KiB\n", p.l2CacheSize >> 10);
    printf("  Shared memory per SM: %ld KiB\n", p.sharedMemPerMultiprocessor >> 10);
    printf("  Shared memory per block: %ld KiB\n", p.sharedMemPerBlock >> 10);
    printf("  Shared memory per block opt-in: %ld KiB\n", p.sharedMemPerBlockOptin >> 10);
    printf("  Registers per SM: %d\n", p.regsPerMultiprocessor);
    printf("  Registers per block: %d\n", p.regsPerBlock);
    printf("  Single:Double performance ratio: %d:1\n", p.singleToDoublePrecisionPerfRatio);
}



int main(int argc, char ** argv)
{
    int device = -1;
    int device_count;

    if(argc > 1)
        device = atoi(argv[1]);

    hipGetDeviceCount(&device_count);

    printf("There are %d cuda-capable devices detected\n\n", device_count);

    if(device >= 0)
    {
        if(device < device_count)
        {
            print_properties_for_gpu(device);
        }
        else
        {
            printf("Invalid device index\n");
        }
    }
    else
    {
        for(device = 0; device < device_count; device++)
        {
            if(device != 0)
                printf("\n");
            print_properties_for_gpu(device);
        }
    }

    return 0;
}
